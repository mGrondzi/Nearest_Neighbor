#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdio>
#include <iostream>
#include <IL/il.h>
#include "image.h"

#define FACTOR 16

__global__ void nearest_neighbor(uint32_t *dp, uint32_t *sp, ILuint width, int factor, int maxThreads)
{
	int i = threadIdx.x;
	i = i +  maxThreads * blockIdx.x;

	int row = i / width;
	int col = i % width;

	dp[i] = sp[((row / factor) * width/factor + (col / factor))];
}

extern void CallNearestNeighbor();

int main()
{

	//CallNearestNeighbor();
	ILuint handle;
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilGenImages(1, &handle);
	ilBindImage(handle);
	hipError_t cudaStatus;
	hipDeviceProp_t deviceProp;

	hipGetDeviceProperties(&deviceProp, 0);

	std::cout << "Read images" << std::endl;
	Image *org = new Image("image4.png");
	Image *res = new Image("image5.png", org->getWidth() * FACTOR, org->getHeight() * FACTOR);

	std::cout << "Execute nearest neighbor algorithm" << std::endl;
	
	// prepare
	uint32_t *input, *out;

	cudaStatus = hipSetDevice(0);
	hipMalloc(&input, org->getHeight() * org->getWidth() * sizeof(uint32_t));
	hipMalloc(&out, res->getHeight() * res->getWidth() * sizeof(uint32_t));
	
	hipMemcpy(input, org->getData(), org->getHeight() * org->getWidth() * sizeof(uint32_t), hipMemcpyHostToDevice);
	
	nearest_neighbor<<<(res->getHeight() * res->getWidth()) / deviceProp.maxThreadsPerBlock, deviceProp.maxThreadsPerBlock>>>(out, input, res->getWidth(), FACTOR, deviceProp.maxThreadsPerBlock);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	cudaStatus = hipMemcpy(res->getData(), out, res->getHeight() * res->getWidth() * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	std::cout << "Save new image" << std::endl;
	res->saveImage();

	std::cout << "Free space" << std::endl;
	delete org;
	delete res;
	ilDeleteImages(1, &handle);

	hipFree(input);
	hipFree(out);

	std:getchar();

    return 0;
}

