#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdio>
#include <iostream>
#include <IL/il.h>
#include "image.h"

#define FACTOR 4

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

/*__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

	// Here our shit.
}*/

__global__ void nearest_neighbor(uint32_t *dp, uint32_t *sp, ILuint height, ILuint width, int factor)
{
	for (int row = 0; row < height; ++row) {
		for (int col = 0; col < width; ++col) {
			dp[(row * width + col)] = sp[((row / factor) * width/factor + (col / factor))];
		}
	}
}

extern void CallNearestNeighbor();

int main()
{

	//CallNearestNeighbor();
	ILuint handle;
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilGenImages(1, &handle);
	ilBindImage(handle);
	hipError_t cudaStatus;


	std::cout << "Read images" << std::endl;
	Image *org = new Image("image.png");
	Image *res = new Image("image3.png", org->getWidth() * FACTOR, org->getHeight() * FACTOR);

	std::cout << "Execute nearest neighbor algorithm" << std::endl;
	
	// prepare
	uint32_t *input, *out, *temp;
	cudaStatus = hipSetDevice(0);
	hipMalloc(&input, org->getHeight() * org->getWidth() * sizeof(uint32_t));
	hipMalloc(&out, res->getHeight() * res->getWidth() * sizeof(uint32_t));
	
	hipMemcpy(input, org->getData(), org->getHeight() * org->getWidth() * sizeof(uint32_t), hipMemcpyHostToDevice);
	

	nearest_neighbor<<<1,1>>>(out, input, res->getHeight(), res->getWidth(), 4);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}


	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	cudaStatus = hipMemcpy(res->getData(), out, res->getHeight() * res->getWidth() * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	std::cout << "Save new image" << std::endl;
	res->saveImage();

	std::cout << "Free space" << std::endl;
	delete org;
	delete res;
	ilDeleteImages(1, &handle);

	hipFree(input);
	hipFree(out);

    return 0;
}

